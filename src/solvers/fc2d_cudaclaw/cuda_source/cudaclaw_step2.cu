#include "../fc2d_cudaclaw.h"
#include "cudaclaw_allocate.h"
#include "cudaclaw_update_q.h"
#include "cudaclaw_flux2.h"

#include <fc2d_cudaclaw_options.h>

#include <fclaw2d_patch.h>
#include <fclaw2d_global.h>
#include <fclaw2d_clawpatch.h>
#include <fclaw2d_clawpatch_options.h>

#include "../fc2d_cudaclaw_check.cu"  /* CHECK defined here */

#include <hipblas.h>
#include <fc2d_cuda_profiler.h>
#include <hipcub/hipcub.hpp>

static double* s_membuffer;
static double* s_membuffer_dev;

cudaclaw_fluxes_t* s_array_fluxes_struct_dev;


void cudaclaw_allocate_buffers(fclaw2d_global_t *glob)
{
    fclaw2d_clawpatch_options_t *clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    int mx = clawpatch_opt->mx;
    int my = clawpatch_opt->my;
    int mbc = clawpatch_opt->mbc;
    int maux = clawpatch_opt->maux;
    int meqn = clawpatch_opt->meqn;  

    int batch_size = FC2D_CUDACLAW_BUFFER_LEN;
    size_t size = (2*mbc+mx)*(2*mbc+my);
    size_t bytes = batch_size*size*(meqn + maux)*sizeof(double);

    CHECK(hipHostMalloc((void**)&s_membuffer,bytes));    
    CHECK(hipMalloc((void**)&s_membuffer_dev, bytes)); 

    CHECK(hipMalloc(&s_array_fluxes_struct_dev, batch_size*sizeof(cudaclaw_fluxes_t)));
}

void cudaclaw_deallocate_buffers(fclaw2d_global_t *glob)
{
    hipHostFree(s_membuffer);
    hipFree(s_membuffer_dev);
    hipFree(s_array_fluxes_struct_dev);
}


double cudaclaw_step2_batch(fclaw2d_global_t *glob,
        cudaclaw_fluxes_t* array_fluxes_struct, 
        int batch_size, double t, double dt)
{
    PROFILE_CUDA_GROUP("cudaclaw_step2_batch",5);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //float milliseconds;
    int i;

    double maxcfl = 0.0;

    FCLAW_ASSERT(batch_size !=0);

    /* To get patch-independent parameters */
    fc2d_cudaclaw_options_t *clawopt;
    fclaw2d_clawpatch_options_t *clawpatch_opt;

    clawopt = fc2d_cudaclaw_get_options(glob);
    int mwaves = clawopt->mwaves;

    fc2d_cudaclaw_vtable_t*  cuclaw_vt = fc2d_cudaclaw_vt();
    FCLAW_ASSERT(cuclaw_vt->cuda_rpn2 != NULL);


    clawpatch_opt = fclaw2d_clawpatch_get_options(glob);
    int mx = clawpatch_opt->mx;
    int my = clawpatch_opt->my;
    int mbc = clawpatch_opt->mbc;
    int maux = clawpatch_opt->maux;
    int meqn = clawpatch_opt->meqn;  

    cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[0]);
    size_t size = batch_size*(fluxes->num + fluxes->num_aux);
    size_t bytes = size*sizeof(double);
    //double *membuffer;
    //double* membuffer_dev;

    /* ---------------------------------- Merge Memory ---------------------------------*/ 
    {
        {
            PROFILE_CUDA_GROUP("Malloc buffer on the host and device",2);    
            FCLAW_ASSERT(s_membuffer != NULL);
            FCLAW_ASSERT(s_membuffer_dev != NULL);
            //CHECK(hipHostMalloc((void**)&membuffer,bytes));

            //CHECK(hipMalloc((void**)&membuffer_dev, bytes));            
        }

        PROFILE_CUDA_GROUP("cudaclaw_copy_loop",3);    
        for(i = 0; i < batch_size; i++)   
        {
            cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);    

            int I_q = i*fluxes->num;
            int I_aux = batch_size*fluxes->num + i*fluxes->num_aux;

            memcpy(&s_membuffer[I_q]  ,fluxes->qold ,fluxes->num_bytes);
            memcpy(&s_membuffer[I_aux],fluxes->aux  ,fluxes->num_bytes_aux);

            /* Assign gpu pointers */
            fluxes->qold_dev = &s_membuffer_dev[I_q];
            fluxes->aux_dev  = &s_membuffer_dev[I_aux];
        }        

        {
            PROFILE_CUDA_GROUP("Copy buffer to device",7);              
            CHECK(hipMemcpy(s_membuffer_dev, s_membuffer, bytes, hipMemcpyHostToDevice));            
        }
    }        


    /* -------------------------------- Work with array --------------------------------*/ 

    //cudaclaw_fluxes_t* array_fluxes_struct_dev = NULL;
    //CHECK(hipMalloc(&array_fluxes_struct_dev, batch_size*sizeof(cudaclaw_fluxes_t)));

    FCLAW_ASSERT(s_array_fluxes_struct_dev != NULL);

    CHECK(hipMemcpy(s_array_fluxes_struct_dev, array_fluxes_struct, 
                     batch_size*sizeof(cudaclaw_fluxes_t), 
                     hipMemcpyHostToDevice));


    dim3 block(128,1,1);
    dim3 grid(1,1,batch_size);

    size_t bytes_per_thread = sizeof(double)*(5*meqn+3*maux+mwaves+meqn*mwaves);
    
    double* maxcflblocks_dev;
    CHECK(hipMalloc(&maxcflblocks_dev,batch_size*sizeof(double))); 
    cudaclaw_flux2_and_update_batch<<<grid,block,128*bytes_per_thread >>>(mx,my,meqn,
                                                                     mbc,maux,mwaves,dt,t,
                                                                     s_array_fluxes_struct_dev,
								                                     maxcflblocks_dev,
                                                                     cuclaw_vt->cuda_rpn2,
                                                                     cuclaw_vt->cuda_b4step2);
    hipDeviceSynchronize();
    CHECK(hipPeekAtLastError());
	
    /* -------------------------------- Finish CFL ------------------------------------*/ 
    {
        PROFILE_CUDA_GROUP("Finish CFL",1);
        void    *temp_storage_dev = NULL;
        size_t  temp_storage_bytes = 0;
        double  *cflgrid_dev;

        hipMalloc(&cflgrid_dev, sizeof(double));  
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMalloc(&temp_storage_dev, temp_storage_bytes);
        CubDebugExit(hipcub::DeviceReduce::Max(temp_storage_dev,temp_storage_bytes,
                                            maxcflblocks_dev,cflgrid_dev,batch_size));
        hipMemcpy(&maxcfl, cflgrid_dev, sizeof(double),hipMemcpyDeviceToHost);
        hipFree(temp_storage_dev);
        hipFree(cflgrid_dev);
    }
    /* ------------------------------ Done with CFL ------------------------------------*/ 

    /* -------------------------- Copy q back to host ----------------------------------*/ 
    CHECK(hipMemcpy(s_membuffer, s_membuffer_dev, batch_size*fluxes->num_bytes, 
                     hipMemcpyDeviceToHost));

    {
        PROFILE_CUDA_GROUP("Copy back to patches loop",2);
        for (int i = 0; i < batch_size; ++i)    
        {      

            cudaclaw_fluxes_t* fluxes = &(array_fluxes_struct[i]);
            int I_q = i*fluxes->num;

            memcpy(fluxes->qold,&s_membuffer[I_q],fluxes->num_bytes);

        }        
    }

    /* ------------------------------ Clean up -----------------------------------------*/ 
    //hipFree(array_fluxes_struct_dev);
    //hipFree(membuffer_dev);
    //hipHostFree(membuffer);

    return maxcfl;
}

